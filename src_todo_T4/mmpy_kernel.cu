#include "hip/hip_runtime.h"
// ;-*- mode: c;-*-
// Matrix multiply device code
#include <assert.h>
#include <math.h>
#include "../src/utils.h"
#include "../src/types.h"
#include "mytypes.h"
using namespace std;

#include <stdio.h>

#if NAIVE == 1
__global__ void matMul(int N, _FTYPE_ *C, _FTYPE_ *A, _FTYPE_ *B) {

    int I =  blockIdx.y*blockDim.y + threadIdx.y;
    int J =  blockIdx.x*blockDim.x + threadIdx.x;
    // printf("blockIdx.y: %d, blockIdx.x %d\n", blockIdx.y, blockIdx.x);
    // printf("blockDim.y: %d, blockDim.x: %d\n", blockDim.y, blockDim.x);
    
    if((I < N) && (J < N)){
        _FTYPE_ _c = 0;
        PRAGMA_UNROLL
        for (unsigned int k = 0; k < N; k++) {
            _FTYPE_ a = A[I * N + k];
            _FTYPE_ b = B[k * N + J];
            _c += a * b;
        }
        C[I * N + J] = _c;
    }
}
#elif BASIC_SHM == 1
//You should be changing the kernel here for the non naive implementation.
__global__ void matMul(int N, _FTYPE_ *C, _FTYPE_ *A, _FTYPE_ *B) {

    extern __shared__ _FTYPE_ sharedMem[]; //use dynamic shared mem
    _FTYPE_ *As = sharedMem;
    _FTYPE_ *Bs = &sharedMem[BLOCK_SIZE * BLOCK_SIZE];

    _FTYPE_ Cij = 0.0;

    int ty = threadIdx.y, tx = threadIdx.x;
    int by = blockIdx.y, bx = blockIdx.x;
    int I = by * BLOCK_SIZE + ty;
    int J = bx * BLOCK_SIZE + tx;

    PRAGMA_UNROLL
    for (int kk = 0; kk < N / BLOCK_SIZE; kk++) {
        if (I < N && J < N) {
            As[ty * BLOCK_SIZE + tx] = A[I * N + kk * BLOCK_SIZE + tx];
            Bs[ty * BLOCK_SIZE + tx] = B[(kk * BLOCK_SIZE + ty) * N + J];
        }
       
        __syncthreads();

        PRAGMA_UNROLL
        for (int k = 0; k < BLOCK_SIZE; k++) {
            Cij += As[ty * BLOCK_SIZE + k] * Bs[k * BLOCK_SIZE + tx];
        }

        __syncthreads();
    }
    if (I < N && J < N) {
        C[I * N + J] = Cij;
    }
    //C[I * N + J] = Cij;
}    

#elif TILING == 1
//You should be changing the kernel here for the non naive implementation.
__global__ void matMul(int N, _FTYPE_ *C, _FTYPE_ *A, _FTYPE_ *B) {

    extern __shared__ _FTYPE_ sharedMem[]; // use dynamic shared memory
    _FTYPE_ *As = sharedMem; // size: TILEDIM_M * TILEDIM_K
    _FTYPE_ *Bs = &sharedMem[TILEDIM_M * TILEDIM_K]; // size: TILEDIM_K * TILEDIM_N

    _FTYPE_ Cij[TILESCALE_M][TILESCALE_N] = {0.0};
    #if PRE_LOAD_CIJ == 1
    register _FTYPE_ Ci_reg[TILESCALE_M] = {0.0};
    register _FTYPE_ Cj_reg[TILESCALE_N] = {0.0};
    #endif

    const int tx = threadIdx.x, ty = threadIdx.y;
    const int bx = blockIdx.x, by = blockIdx.y;
    const int I = (by * blockDim.y + ty) * TILESCALE_M;
    const int J = (bx * blockDim.x + tx) * TILESCALE_N;

    const int threadID = threadIdx.y * blockDim.x + threadIdx.x;

    const int tileRowA = threadID / TILEDIM_K;
    const int tileColA = threadID % TILEDIM_K;
    const int tileRowB = threadID / TILEDIM_N;
    const int tileColB = threadID % TILEDIM_N;

    const int strideA = (BLOCKDIM_X * BLOCKDIM_Y) / TILEDIM_K;
    const int strideB = (BLOCKDIM_X * BLOCKDIM_Y) / TILEDIM_N;

    PRAGMA_UNROLL
    for (int kk = 0; kk < N; kk += TILEDIM_K) {
        // load submatrix A into shared memory
        PRAGMA_UNROLL
        for (int i = 0; i < TILEDIM_M; i += strideA) {
            const int row = tileRowA + TILEDIM_M * by + i;
            const int col = tileColA + kk;
            #if USE_IF_STATEMENT == 0
            As[(tileRowA + i) * TILEDIM_K + tileColA] = (row < N && col < N)? A[row * N + col] : 0;
            #else
            if (row < N && col < N) {
                As[(tileRowA + i) * TILEDIM_K + tileColA] = A[row * N + col];
            }
            #endif
        }

        // load submatrix B into shared memory
        PRAGMA_UNROLL
        for (int i = 0; i < TILEDIM_K; i += strideB) {
            const int row = tileRowB + kk + i;
            const int col = tileColB + TILEDIM_N * bx;
            #if USE_IF_STATEMENT == 0
            Bs[(tileRowB + i) * TILEDIM_N + tileColB] = (row < N && col < N)? B[row * N + col] : 0;
            #else
            if (row < N && col < N) {
                Bs[(tileRowB + i) * TILEDIM_N + tileColB] = B[row * N + col];
            }
            #endif
        }   
        __syncthreads();

        PRAGMA_UNROLL
        for (int k = 0; k < TILEDIM_K; ++k) {
            #if PRE_LOAD_CIJ == 1
            // Preload shared memory into register
            PRAGMA_UNROLL
            for (int i = 0; i < TILESCALE_M; ++i) {
                Ci_reg[i] = As[(TILESCALE_M * ty + i) * TILEDIM_K + k]; // As[TILESCALE_M * ty + i][k]
            }

            PRAGMA_UNROLL
            for (int i = 0; i < TILESCALE_N; ++i) {
                Cj_reg[i] = Bs[k * TILEDIM_N + TILESCALE_N * tx + i]; // Bs[k][TILESCALE_N * tx + i]
            }

            // Calculate submatrix C
            PRAGMA_UNROLL
            for (int i = 0; i < TILESCALE_M; ++i) {
                PRAGMA_UNROLL
                for (int j = 0; j < TILESCALE_N; ++j) {
                    Cij[i][j] += Ci_reg[i] * Cj_reg[j];
                }
            }
            #else
            // Calculate submatrix C
            PRAGMA_UNROLL
            for (int i = 0; i < TILESCALE_M; ++i) {
                PRAGMA_UNROLL
                for (int j = 0; j < TILESCALE_N; ++j) {

                    Cij[i][j] += As[(TILESCALE_M * ty + i) * TILEDIM_K + k] 
                                * Bs[k * TILEDIM_N + TILESCALE_N * tx + j];
                }
            } 
            #endif
        }

        __syncthreads();
    }

    // Calculate C
    PRAGMA_UNROLL
    for (int i = 0; i < TILESCALE_M; i++) {
        PRAGMA_UNROLL
        for (int j = 0; j < TILESCALE_N; j++) {
            const int row = I + i;
            const int col = J + j;
            if (row < N && col < N) {
                C[row * N + col] = Cij[i][j];
            }
            //C[row * N + col] = Cij[i][j];
        }
    }
}
#endif